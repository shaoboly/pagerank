#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h> 

#include "hipsparse.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <>
#include <sstream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

using namespace std;

//#pragma comment(lib,"cusparse.lib")

#define dynamic_max 256
#define dynamic_block_size 256
#define max_bin 12

#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError(const char* file, const int32_t line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		std::cerr << "cudaCheckError() failed at " << file << ":" << line << " : "
			<< hipGetErrorString(err) << endl;
		exit(-1);
	}
}


__global__ void init_pagerank_d(int n ,float *pagerank_d)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n) {
		//pagerank_d[i] = 1.0 / (float)n_vertices;
		pagerank_d[i] = 1.0;
	}
}

//�Ȳ�Ӧ��
__global__ void init_pagerank_d_by_out(int n, float *pagerank_d, int *destination_offsets_d)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n) {
		//pagerank_d[i] = 1.0 / (float)n_vertices;
		int out_d = destination_offsets_d[i + 1] - destination_offsets_d[i];
		if (out_d == 0)
			pagerank_d[i] = 1.0 / n;
		else
			pagerank_d[i] = 1.0 / out_d;
	}
}


__global__ void get_dangling_value(int n, float *pagerank_d, float *bookmark_d, float *damping_value_d)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n && bookmark_d[i] >0.1) {
		atomicAdd(damping_value_d, bookmark_d[i] * pagerank_d[i]);
	}
}

__global__ void get_dangling_value_v1(int dangling_nnz, float *pagerank_d, int *x_idx_d, float *damping_value_d)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < dangling_nnz) {
		atomicAdd(damping_value_d, 1.0 * pagerank_d[x_idx_d[i]]);
	}
}

//��������ͣ����ۼӵ�ֵ����ȥ������ͣ�����Ϊ1/n
__global__ void finalPagerankArrayForIteration(float * pagerank_next_d, int n_vertices, float dangling_value_h,float alpha) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;


	if (i < n_vertices) {
		//pagerank_next_d[i] += (dangling_value2 + (1 - 0.85)) / ((float)n_vertices);
		pagerank_next_d[i] += dangling_value_h* 1.0 / (float)n_vertices + 1 - alpha;
	}
}


__global__ void child_kernal(int parant_idx, int begin, int in_d, float alpha, float *weights_d, int *destination_offsets_d, int *source_indices_d, float *pagerank_d, float *pagerank_next_d)
{
	extern __shared__ float storage[];
	int cidx = threadIdx.x;
	int idx = cidx + blockDim.x * blockIdx.x;
	int t_n = blockDim.x * gridDim.x;
	float tmp = 0;
	while (idx < in_d)
	{
		tmp += pagerank_d[source_indices_d[begin + idx]] * weights_d[begin + idx] * alpha;
		idx += t_n;
	}
	storage[cidx] = tmp;
	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cidx < i)
		{
			storage[cidx] += storage[cidx + i];
		}
		__syncthreads();
		i >>= 1;
	}
	if (cidx == 0)
		atomicAdd(&pagerank_next_d[parant_idx], storage[0]);

}

__global__ void dynamic_test_big_node_v1(int bin_n, int child_block_size, int *bin, float alpha, float *weights_d, int *destination_offsets_d, int *source_indices_d, float *pagerank_d, float *pagerank_next_d, float remain_value)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx < bin_n){
		int now_node = bin[idx];
		pagerank_next_d[now_node] += remain_value;
		int begin = destination_offsets_d[now_node];
		int end = destination_offsets_d[now_node + 1];
		int in_d = end - begin;


		child_kernal << <1, child_block_size, child_block_size*sizeof(float) >> >(now_node, begin, in_d, alpha, weights_d, destination_offsets_d, source_indices_d, pagerank_d, pagerank_next_d);
	}
}

__global__ void dynamic_test_big_node_v2(int bin_n, int child_block_size, int *bin, float alpha, float *weights_d, int *destination_offsets_d, int *source_indices_d, float *pagerank_d, float *pagerank_next_d, float remain_value)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx < bin_n){
		int now_node = bin[idx];
		pagerank_next_d[now_node] += remain_value;
		int begin = destination_offsets_d[now_node];
		int end = destination_offsets_d[now_node + 1];
		int in_d = end - begin;

		int thread_num = in_d/4;

		int numOfBlocks = 1;                          // default example value for 1000 vertex graph
		int threadsPerBlock = 1;                   // default example value for 1000 vertex graph

		if (thread_num <= 1024) {
			threadsPerBlock = child_block_size;
			numOfBlocks = 1;
		}
		else {
			threadsPerBlock = 1024;
			numOfBlocks = (thread_num + 1023) / 1024;   // The "+ 1023" ensures we round up
		}

		child_kernal << <numOfBlocks, threadsPerBlock, threadsPerBlock*sizeof(float) >> >(now_node, begin, in_d, alpha, weights_d, destination_offsets_d, source_indices_d, pagerank_d, pagerank_next_d);
	}
}


__global__ void addToNextPagerankArray(int n, int nnz, float alpha, float *weights_d, int *destination_offsets_d, int *source_indices_d, float *pagerank_d, float *pagerank_next_d, float remain_value)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < n){
		pagerank_next_d[idx] += remain_value;
		int begin = destination_offsets_d[idx];
		int end = destination_offsets_d[idx + 1];
		int in_d = end - begin;

		if (in_d >= dynamic_max)
		{
			child_kernal << <1, dynamic_block_size >> >(idx, begin, in_d, alpha, weights_d, destination_offsets_d, source_indices_d, pagerank_d, pagerank_next_d);
		}
		else
		{
			for (; begin < end; begin++){
				pagerank_next_d[idx] += pagerank_d[source_indices_d[begin]] * weights_d[begin] * alpha;
			}
		}
	}
	//__syncthreads();
}

__global__ void dynamic_test(int bin_n, int nnz,int *bin, float alpha, float *weights_d, int *destination_offsets_d, int *source_indices_d, float *pagerank_d, float *pagerank_next_d, float remain_value)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;


	if (idx < bin_n){
		int now_node = bin[idx];
		pagerank_next_d[now_node] = 0;
		pagerank_next_d[now_node] += remain_value;
		int begin = destination_offsets_d[now_node];
		int end = destination_offsets_d[now_node + 1];
		for (; begin < end; begin++){
			pagerank_next_d[now_node] += pagerank_d[source_indices_d[begin]] * weights_d[begin] * alpha;
		}
	}
}

__global__ void convergence(float * pagerank_d, float * pagerank_next_d, float * reduced_sums_d, int n_vertices) {
	// Each thread computes the diff for two vertexes (thus, half # of blocks needed for this function)
	// Because of this, we need to handle the case where only one block is needed
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i_thr = threadIdx.x;

	__shared__ float sums[1024];                       // blockDim.x == 1024

	float temp1, temp2;

	if (i < 1024) {
		reduced_sums_d[i] = 0;
	}

	if (i < n_vertices) {
		temp1 = pagerank_next_d[i] - pagerank_d[i];
		if (i + (1024 * gridDim.x) < n_vertices) {
			temp2 = pagerank_next_d[i + (1024 * gridDim.x)] - pagerank_d[i + (1024 * gridDim.x)];
		}
		else{
			temp2 = 0;
		}

		if (temp1 < 0) {
			temp1 = temp1 * (-1);
		}
		if (temp2 < 0) {
			temp2 = temp2 * (-1);
		}

		sums[i_thr] = temp1 + temp2;
	}
	else {
		sums[i_thr] = 0;
	}
	__syncthreads();

	int j, index, index2;
	index = i_thr;

	//���Ż�
	for (j = 0; j < 10; j++) {                    // 10 times as 2^10 = 1024 threads
		if ((index + 1) % (2 * (1 << j)) == 0) {    // Note: 1 << j == 2^j
			index2 = index - (1 << j);
			sums[index] += sums[index2];
		}
		__syncthreads();
	}

	reduced_sums_d[blockIdx.x] = sums[1023];
}

__global__ void getConvergence(float * reduced_sums_d, float * diff) {
	int j, index, index2;
	index = threadIdx.x;

	for (j = 0; j < 10; j++) {                    // 10 times as 2^10 = 1024 threads
		if ((index + 1) % (2 * (1 << j)) == 0) {    // Note: 1 << j == 2^j
			index2 = index - (1 << j);
			reduced_sums_d[index] += reduced_sums_d[index2];
		}
		__syncthreads();
	}

	*diff = reduced_sums_d[1023];
}

void str2int(int &int_temp, const string &string_temp)
{
	stringstream stream(string_temp);
	stream >> int_temp;
}

void compress_dangling_node_vector(float *bookmark_h,int n_vertices,int &dangling_nnz, float* &value_vector, int* &x_idx)
{
	dangling_nnz = 0;

	for (int i = 0; i < n_vertices; i++)
	{
		if (bookmark_h[i]>0.5)
			dangling_nnz++;
	}

	value_vector = (float*)malloc(dangling_nnz*sizeof(float));
	x_idx = (int*)malloc(n_vertices*sizeof(int));

	int offset = 0;
	for (int i = 0; i < n_vertices; i++)
	{
		if (bookmark_h[i]>0.5)
		{
			x_idx[offset] = i;
			value_vector[offset++] = bookmark_h[i];
 		}
	}

	return;
}


void initialize(float *cooValHostPtr, int *cooColIndexHostPtr, float *yHostPtr, int *csrRowPtr)
{
	cooValHostPtr[0] = 1.0;
	cooValHostPtr[1] = 2.0;
	cooValHostPtr[2] = 3.0;
	cooValHostPtr[3] = 4.0;
	cooValHostPtr[4] = 5.0;
	cooValHostPtr[5] = 6.0;
	cooValHostPtr[6] = 7.0;
	cooValHostPtr[7] = 8.0;
	cooValHostPtr[8] = 9.0;

	cooValHostPtr[9] = 10.0;

	cooColIndexHostPtr[0] = 0;
	cooColIndexHostPtr[1] = 2;
	cooColIndexHostPtr[2] = 3;
	cooColIndexHostPtr[3] = 1;
	cooColIndexHostPtr[4] = 0;
	cooColIndexHostPtr[5] = 2;
	cooColIndexHostPtr[6] = 3;
	cooColIndexHostPtr[7] = 1;
	cooColIndexHostPtr[8] = 3;

	cooColIndexHostPtr[9] = 0;

	yHostPtr[0] = 10.0;
	yHostPtr[1] = 20.0;
	yHostPtr[2] = 30.0;
	yHostPtr[3] = 40.0;
	/*yHostPtr[4] = 50.0;
	yHostPtr[5] = 60.0;
	yHostPtr[6] = 70.0;
	yHostPtr[7] = 80.0;*/

	csrRowPtr[0] = 0;
	csrRowPtr[1] = 3;
	csrRowPtr[2] = 4;
	csrRowPtr[3] = 7;
	csrRowPtr[4] = 9;

	csrRowPtr[5] = 10;

}


class vertex
{
public:
	int id;
	int in_deg;
	int out_deg;
	vector<int> in_edge;
	vector<int> out_edge;
};


void getInputResultForCsc(vector<vertex> &vertices, string input_name, int &n, int &nnz, int *&destination_offsets_h, int *&source_indices_h, float *&weights_h, float *&bookmark_h)
{
	ifstream input_file(input_name);
	int tmp_va, tmp_vb;

	input_file >> n >> nnz;

	destination_offsets_h = (int*)malloc((n + 1)*sizeof(int));
	source_indices_h = (int*)malloc(nnz*sizeof(int));
	weights_h = (float*)malloc(nnz*sizeof(float));
	bookmark_h = (float*)malloc(n*sizeof(float));

	for (int i = 0; i < n; i++)
	{
		vertex tmp_v;
		tmp_v.id = i;
		tmp_v.in_deg = 0;
		tmp_v.out_deg = 0;
		vertices.push_back(tmp_v);
	}

	for (int i = 0; i < nnz; i++)
	{
		input_file >> tmp_va >> tmp_vb;
		tmp_va--;
		tmp_vb--;
		vertices[tmp_va].out_deg += 1;
		vertices[tmp_va].out_edge.push_back(tmp_vb);

		vertices[tmp_vb].in_deg += 1;
		vertices[tmp_vb].in_edge.push_back(tmp_va);
	}

	//cout << "read graph success!" << endl;
	int tmp_offset = 0;
	for (int i = 0; i < n; i++)
	{
		destination_offsets_h[i] = tmp_offset;

		if (vertices[i].out_deg == 0)
			bookmark_h[i] = 1.0f;
		else
			bookmark_h[i] = 0.0f;

		for (int j = 0; j < vertices[i].in_deg; j++){
			source_indices_h[tmp_offset + j] = vertices[i].in_edge[j];
			weights_h[tmp_offset + j] = (float)1.0 / (float)vertices[source_indices_h[tmp_offset + j]].out_deg;
		}
		tmp_offset += vertices[i].in_deg;

	}
	destination_offsets_h[n] = tmp_offset;
}


vector<vector<int>> binary_resort(vector<vertex> &vertices, int n, int nnz, int &bins_number)
{
	int max_label = 0;
	vector<vector<int>> all_b(max_bin);

	for (int i = 0; i < n; i++)
	{
		if (vertices[i].in_deg == 0)
		{
			all_b[0].push_back(i);
			continue;
		}
		int now_label = (unsigned int)log2(double(vertices[i].in_deg) - 0.1) + 1;

		if (now_label >= max_bin){
			all_b[max_bin - 1].push_back(i);
			max_label = max_bin - 1;
			continue;
		}

		all_b[now_label].push_back(i);

		if (max_label < now_label)
			max_label = now_label;
	}
	for (int i = 0; i < max_bin; i++)
	{
		cout << i << " " << all_b[i].size() << endl;
	}
	bins_number = max_label+1;
	
	return all_b;
}


//transform to bins
void trans_to_b(vector<vertex> vertices, int n, int nnz, int* &g1, vector<int*> &g2, int &bins_number, vector<vector<int>> &all_bin, int &dp_max_num, int &row_max)
{
	row_max = 0;
	for (int i = 0; i < dp_max_num; i++)
	{
		row_max += all_bin[i].size();
	}

	g1 = (int*)malloc((row_max)*sizeof(int));

	int tmp_ind = 0;
	for (int i = 0; i < dp_max_num; i++)
	{
		for (int j = 0; j < all_bin[i].size(); j++)
		{
			g1[tmp_ind++] = all_bin[i][j];
		}
	}

	for (int i = dp_max_num; i < bins_number; i++)
	{
		int *tmp = (int*)malloc(all_bin[i].size()*sizeof(int));
		for (int j = 0; j < all_bin[i].size(); j++)
		{
			tmp[j] = all_bin[i][j];
		}
		g2.push_back(tmp);
	}

}

void pagerank_cuda(string input_name, string output_name, int max_iteration)
{

	ofstream output_file(output_name);

	//graph imformation
	int  n = 0, nnz = 0, vertex_numsets = 3, edge_numsets = 1;
	int *destination_offsets_h = NULL, *source_indices_h = NULL;
	int *destination_offsets_d, *source_indices_d;
	float *weights_h = NULL, *bookmark_h = NULL;
	float *weights_d,*bookmark_d;
	float dampling_value_h;
	float *damping_value_d;

	float *reduced_sums_d;
	float * d_diff;
	hipMalloc((void **)&reduced_sums_d, 1024 * sizeof(float));
	hipMalloc((void **)&d_diff, sizeof(float));


	float alpha = 1;
	float beta = 0;
	//void** vertex_dim;

	vector<vertex> vertices;
	getInputResultForCsc(vertices,input_name, n, nnz, destination_offsets_h, source_indices_h, weights_h, bookmark_h);


	//to sort in bins
	int bins_number = 0;
	vector<vector<int>> all_bin = binary_resort(vertices, n, nnz, bins_number);

	int dp_max_num = 8;
	int row_max = 0;
	
	int *g1 = NULL;
	vector<int*> g2;
	trans_to_b(vertices, n, nnz, g1, g2, bins_number, all_bin, dp_max_num, row_max);
	cout << "1" << endl;
	//--------------------------//
	
	int *g1_device;
	hipMalloc((void **)&g1_device, row_max*sizeof(int));
	
	hipMemcpy(g1_device, g1, row_max*sizeof(int), hipMemcpyHostToDevice);
	cout << "g1:" << row_max << endl;

	int g2_size = bins_number - dp_max_num;
	vector<int *> device_g2;

	cout << g2_size << endl;
	int* g2_b_size = new int[g2_size];
	for (int i = 0; i < g2_size; i++)
	{
		g2_b_size[i] = all_bin[i + dp_max_num].size();
		int *d_tmp;
		hipMalloc((void **)&d_tmp, g2_b_size[i] * sizeof(int));
		hipMemcpy(d_tmp, g2[i], g2_b_size[i] * sizeof(int), hipMemcpyHostToDevice);
		cout << "g" << i + dp_max_num << " " << g2_b_size[i] << endl;
		device_g2.push_back(d_tmp);

	}





	int g1_block_number, g1_thread_per_block;
	if (row_max <= 1024) {
		g1_thread_per_block = row_max;
		g1_block_number = 1;
	}
	else {
		g1_thread_per_block = 1024;
		g1_block_number = (row_max + 1023) / 1024;   // The "+ 1023" ensures we round up
	}

	vector<int> g2_block_number, g2_thread_per_block;

	int child_thread_per_block = 4;
	for (int i = 0; i < g2_size; i++)
	{
		int tmp_block_number = (g2_b_size[i] + child_thread_per_block-1) / child_thread_per_block, tmp_thread_per_block = child_thread_per_block;
		
		/*if (g2_b_size[i] <= 1024) {
			tmp_thread_per_block = g2_b_size[i];
			tmp_block_number = 1;
		}
		else {
			tmp_thread_per_block = 1024;
			tmp_block_number = (g2_b_size[i] +1023) / 1024;   // The "+ 1023" ensures we round up
		}*/

		g2_block_number.push_back(tmp_block_number);
		g2_thread_per_block.push_back(tmp_thread_per_block);

		//g2_block_number.push_back(tmp_block_number);

	}

	int size[] = { 128, 256, 512, 1024, 1024,1024,1024,1024 };
	//---------------------------//


	float *pagerank_d, *pagerank_next_d, *pagerank_h = (float*)malloc(n*sizeof(float));

	//cuSparse parameters useless thie version
	hipsparseHandle_t handle;
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);//shape of matrix(triangle or )
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);//index 0 or 1
	hipsparseCreate(&handle);

	hipMalloc((void **)&weights_d, nnz*sizeof(float));
	hipMalloc((void **)&bookmark_d, n*sizeof(float));
	hipMalloc((void **)&pagerank_d, n * sizeof(float));
	hipMalloc((void **)&pagerank_next_d, n * sizeof(float));
	hipMalloc((void **)&destination_offsets_d, (n + 1)*sizeof(int));
	hipMalloc((void **)&source_indices_d, nnz*sizeof(int));
	hipMalloc((void **)&damping_value_d, sizeof(float));


	int n_iterations = 3000;
	int iteration = 0;
	int numOfBlocks = 1;                          // default example value for 1000 vertex graph
	int threadsPerBlock = 1000;                   // default example value for 1000 vertex graph

	if (n <= 1024) {
		threadsPerBlock = n;
		numOfBlocks = 1;
	}
	else {
		threadsPerBlock = 1024;
		numOfBlocks = (n + 1023) / 1024;   // The "+ 1023" ensures we round up
	}


	//initialize origin pagerank sum, should be 1 or n
	/*for (int i = 0; i < n; i++)
	{
		pagerank_h[i] = 1;
	}*/


	// Error code to check return values for CUDA calls
	hipFree(0);   // Set the cuda context here so that when we time, we're not including initial overhead
	hipError_t err = hipSuccess;

	//initialized
	init_pagerank_d << <numOfBlocks, threadsPerBlock >> >(n, pagerank_d);
	hipDeviceSynchronize();

	hipMemcpy(weights_d, weights_h, nnz*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bookmark_d, bookmark_h, n*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(pagerank_d, pagerank_h, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(destination_offsets_d, destination_offsets_h, (n + 1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(source_indices_d, source_indices_h, nnz*sizeof(int), hipMemcpyHostToDevice);

	alpha = 0.85;
	float epsilon = 0.000001 * n;
	float h_diff = epsilon + 1;
	int n_blocks = (n + 2048 - 1) / 2048;
	if (n_blocks == 0){
		n_blocks = 1;
	}

	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);
	sdkResetTimer(&hTimer);

	float remain_value;

	//value about dangling nodes
	float *value_vector_h = NULL, *value_vector_d;
	int *x_idx_h = NULL,*x_idx_d;
	int dangling_nnz = 0;
	compress_dangling_node_vector(bookmark_h, n, dangling_nnz, value_vector_h, x_idx_h);
	
	cout << dangling_nnz << endl;

	hipMalloc((void **)&value_vector_d, dangling_nnz*sizeof(float));
	hipMalloc((void **)&x_idx_d, dangling_nnz*sizeof(int));

	hipMemcpy(value_vector_d, value_vector_h, dangling_nnz*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(x_idx_d, x_idx_h, dangling_nnz*sizeof(int), hipMemcpyHostToDevice);

	//---------------------------
	float base_time = 0;
	float dangling_time = 0;
	float pull_time = 0;
	float g1_time = 0;
	float g2_time = 0;
	//-------------------------------

	hipStream_t stream[15];
	for (int i = 0; i < 15; ++i) 
		hipStreamCreate(&stream[i]);


	for (iteration = 0; iteration < max_iteration; iteration++)
	{
		//hipMemset((void *)z, 0, 2 * (m)*sizeof(float));

		base_time = sdkGetTimerValue(&hTimer);

		//if (i < 10 || i % 5 == 0)
		//{
		    //dangling_sum to 0
			dampling_value_h = 0;
			//err = hipMemcpy(damping_value_d, &dampling_value_h, sizeof(float), hipMemcpyHostToDevice);


			hipDeviceSynchronize();
			sdkStartTimer(&hTimer);
			hipsparseSdoti(handle, dangling_nnz, value_vector_d, x_idx_d, pagerank_d, &dampling_value_h, HIPSPARSE_INDEX_BASE_ZERO);
			//get_dangling_value << <numOfBlocks, threadsPerBlock >> >(n, pagerank_d, bookmark_d, damping_value_d);
			//get_dangling_value_v1 << <numOfBlocks, threadsPerBlock >> >(dangling_nnz, pagerank_d, x_idx_d, damping_value_d);
			hipDeviceSynchronize();
			sdkStopTimer(&hTimer);


			dangling_time += sdkGetTimerValue(&hTimer) - base_time;
			base_time = sdkGetTimerValue(&hTimer);


			//err = hipMemcpy(&dampling_value_h, damping_value_d, sizeof(float), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();

			dampling_value_h *= alpha;
			//cout << dampling_value_h << endl;
		//}

		
		
		remain_value = dampling_value_h* 1.0 / (float)n + 1 - alpha;
		//hipMemcpy(damping_value_d, &dampling_value_h, sizeof(float), hipMemcpyHostToDevice);
		hipMemset((void *)pagerank_next_d, 0, (n)*sizeof(float));

		hipDeviceSynchronize();
		sdkStartTimer(&hTimer);
		

		//addToNextPagerankArray << <numOfBlocks, threadsPerBlock >> >(n, nnz, alpha, weights_d, destination_offsets_d, source_indices_d, pagerank_d, pagerank_next_d, remain_value);
		//hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &alpha, descr, weights_d, destination_offsets_d, source_indices_d, pagerank_d, &beta, pagerank_next_d);
		

		//g2--------------------------------------------------------------------------------//

		//g2_size--;
		for (int j = 0; j < g2_size; j++)
		{
			dynamic_test_big_node_v2 << <g2_block_number[j], g2_thread_per_block[j], 0, stream[j] >> >(g2_b_size[j], size[j], device_g2[j], alpha, weights_d, destination_offsets_d, source_indices_d, pagerank_d, pagerank_next_d, remain_value);
		}
		
		//
		//int j = g2_size;
		//dynamic_test_big_node_v2 << <g2_block_number[j], g2_thread_per_block[j],0, stream[j]>> >(g2_b_size[j], size[j], device_g2[j], alpha, weights_d, destination_offsets_d, source_indices_d, pagerank_d, pagerank_next_d, remain_value);


		//-----------------------------------//

		//-=------------------------------//ִ��g1

		
		dynamic_test << <g1_block_number, g1_thread_per_block, 0, stream[14] >> >(row_max, nnz, g1_device, alpha, weights_d, destination_offsets_d, source_indices_d, pagerank_d, pagerank_next_d, remain_value);
		
		//hipDeviceSynchronize();
		//sdkStopTimer(&hTimer);

		//g1_time += sdkGetTimerValue(&hTimer) - base_time;
		//base_time = sdkGetTimerValue(&hTimer);
		//---------------------------------//

		//sdkStartTimer(&hTimer);

		
		hipDeviceSynchronize();
		//hipDeviceSynchronize();

		sdkStopTimer(&hTimer);
		//g2_time += sdkGetTimerValue(&hTimer) - base_time;
		//base_time = sdkGetTimerValue(&hTimer);

		pull_time += sdkGetTimerValue(&hTimer) - base_time;
		base_time = sdkGetTimerValue(&hTimer);


		
		
		hipDeviceSynchronize();
		// Test for convergence
		//hipDeviceSynchronize();


		sdkStartTimer(&hTimer);
		convergence << <n_blocks, 1024 >> >(pagerank_d, pagerank_next_d, reduced_sums_d, n);
		getConvergence << <1, 1024 >> >(reduced_sums_d, d_diff);
		hipDeviceSynchronize();
		sdkStopTimer(&hTimer);

		hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost);
		cout << iteration << " " << h_diff << endl;


		/*sdkStartTimer(&hTimer);

		finalPagerankArrayForIteration <<<numOfBlocks, threadsPerBlock >>>(pagerank_next_d, n, dampling_value_h, alpha);

		hipDeviceSynchronize();
		sdkStopTimer(&hTimer);*/

		hipMemcpy(pagerank_d, pagerank_next_d, n*sizeof(float), hipMemcpyDeviceToDevice);
		//hipMemcpy(pagerank_h, pagerank_next_d, n*sizeof(float), hipMemcpyDeviceToHost);
	}

	printf("Elapsed Time: %.6fms\n", sdkGetTimerValue(&hTimer));

	cout << "dangling_time:" << dangling_time << endl;
	cout << "pull_time:" << pull_time << endl;
	cout << "g1_time: " << g1_time << endl;
	cout << "g2:" << g2_time << endl;

	hipMemcpy(pagerank_h, pagerank_next_d, n*sizeof(float), hipMemcpyDeviceToHost);
	output_file << "Elapsed Time: " << sdkGetTimerValue(&hTimer) << "ms" << endl;

	output_file.setf(ios::fixed);
	output_file.precision(6);
	for (int i = 0; i<n; i++)
	{
		output_file << pagerank_h[i] << endl;
	}


	hipDeviceReset();

	free(pagerank_h);
	free(value_vector_h);
	free(x_idx_h);
	free(destination_offsets_h);
	free(source_indices_h);
	free(weights_h);
	free(bookmark_h);
}
/*
void cuda_sparse()
{


	int m = 5, n = 4, nnz = 10;
	float *cooValHostPtr = new float[nnz];
	//float *zHostPtr = new float[2 * (m)];
	float *zHostPtr = new float[m];

	int *cooColIndexHostPtr = new int[nnz];
	int *csrRowPtr = new int[m + 1];

	int *crsRow, *cooCol;

	float alpha = 1;
	float beta = 0;
	//float *yHostPtr = new float[2 * n];
	float *yHostPtr = new float[n];
	float * y, *cooVal, *z;
	initialize(cooValHostPtr, cooColIndexHostPtr, yHostPtr, csrRowPtr);


	hipsparseHandle_t handle;
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);//������״�����ǡ��ԳƵȣ�
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);//index 0 or 1

	hipsparseCreate(&handle);

	hipMalloc((void **)&cooVal, nnz*sizeof(float));
	hipMalloc((void **)&y, n * sizeof(float));
	hipMalloc((void **)&z, m * sizeof(float));
	hipMalloc((void **)&crsRow, (m + 1)*sizeof(int));
	hipMalloc((void **)&cooCol, nnz*sizeof(int));

	hipMemcpy(cooVal, cooValHostPtr, nnz*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(y, yHostPtr, 2 * n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y, yHostPtr, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(crsRow, csrRowPtr, (m + 1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooCol, cooColIndexHostPtr, nnz*sizeof(int), hipMemcpyHostToDevice);

	//hipMemset((void *)z, 0, 2 * (m)*sizeof(float));
	hipMemset((void *)z, 0, (m)*sizeof(float));

	hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz, &alpha, descr, cooVal, crsRow, cooCol, y, &beta, z);
	//hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, 2, n, nnz, &alpha, descr, cooVal, crsRow, cooCol, y, n, &beta, z, m);

	hipMemcpy(zHostPtr, z,  (m)*sizeof(float), hipMemcpyDeviceToHost);

	//for (int i = 0; i < m; i++)
	//{
	//  //if(i%(2)==0&&i!=0)
	//  //  cout<<endl;
	//  cout<<zHostPtr[i]<<" "<<zHostPtr[i+m]<<endl;
	//}
	for (int i = 0; i<m; i++)
	{
		cout << zHostPtr[i] << " ";
	}
}*/

int main(int argc, char **argv)
{
	string input_name = "web-Stanford.txt";
	string output_name = "output.txt";

	int max_iteration = 40;

	if (argc > 1)
		input_name = argv[1];
	if (argc > 2)
		output_name = argv[2];
	if (argc > 3)
		str2int(max_iteration, argv[3]);

	//cuda_sparse();
	pagerank_cuda(input_name, output_name, max_iteration);
	return 0;
}